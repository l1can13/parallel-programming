
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <hiprand/hiprand.h>
#include <hipblas.h>
#include <iostream>

//GPU_fill_rand() - ������� ��������� ��������� �������
//gpu_blas_mmul() - ������� ��������� ������
//print_matrix() - ������� ������ �������

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(float* A, int nr_rows_A, int nr_cols_A) {
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// Fill the array with random numbers on the device
	hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const float* A, const float* B, float* C, const int m, const int k, const int n) {
	int lda = m, ldb = k, ldc = m;
	const float alf = 1;
	const float bet = 0;
	const float* alpha = &alf;
	const float* beta = &bet;
	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	// Destroy the handle
	hipblasDestroy(handle);
}

void print_matrix(const float* A, int nr_rows_A, int nr_cols_A) {
	for (int i = 0; i < nr_rows_A; ++i) {
		for (int j = 0; j < nr_cols_A; ++j) {
			std::cout << A[i * nr_cols_A + j] << " ";
		}
		std::cout << std::endl;
	}
}

int main() {
	// Allocate 3 arrays on CPU
	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;
	// for simplicity we are going to use square arrays
	nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 3;
	float* h_A = (float*)malloc(nr_rows_A * nr_cols_A * sizeof(float));
	float* h_B = (float*)malloc(nr_rows_B * nr_cols_B * sizeof(float));
	float* h_C = (float*)malloc(nr_rows_C * nr_cols_C * sizeof(float));
	// Allocate 3 arrays on GPU
	float* d_A, * d_B, * d_C;
	hipMalloc(&d_A, nr_rows_A * nr_cols_A * sizeof(float));
	hipMalloc(&d_B, nr_rows_B * nr_cols_B * sizeof(float));
	hipMalloc(&d_C, nr_rows_C * nr_cols_C * sizeof(float));
	// Fill the arrays A and B on GPU with random numbers
	GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
	GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);
	// Optionally we can copy the data back on CPU and print the arrays
	hipMemcpy(h_A, d_A, nr_rows_A * nr_cols_A * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_B, d_B, nr_rows_B * nr_cols_B * sizeof(float), hipMemcpyDeviceToHost);
	std::cout << "A =" << std::endl;
	print_matrix(h_A, nr_rows_A, nr_cols_A);
	std::cout << "B =" << std::endl;
	print_matrix(h_B, nr_rows_B, nr_cols_B);
	// Multiply A and B on GPU
	gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);
	// Copy (and print) the result on host memory
	hipMemcpy(h_C, d_C, nr_rows_C * nr_cols_C * sizeof(float), hipMemcpyDeviceToHost);
	std::cout << "C =" << std::endl;
	print_matrix(h_C, nr_rows_C, nr_cols_C);

	//Free GPU memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// Free CPU memory
	free(h_A);
	free(h_B);
	free(h_C);

	getchar();

	return 0;
}