
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void addKernel(int* c, int* a, int* b, unsigned int size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    for (; index < size; index += (blockDim.x * gridDim.x)) {
        c[index] = a[index] + b[index];
    }
}

#define kernel addKernel
#define GRID_SIZE 1024
#define BLOCK_SIZE 1024
#define N 7600000


int main(int argc, char* argv[])
{

    int n = N;

    printf("n = %d\n", n);

    int n2b = n * sizeof(int);
    int n2 = n;

    // ��������� ������ �� �����
    int* a = (int*)calloc(n2, sizeof(int));
    int* b = (int*)calloc(n2, sizeof(int));
    int* c = (int*)calloc(n2, sizeof(int));
    // ������������� ��������
    for (int i = 0; i < n; i++) {
        a[i] = 1;
        b[i] = 1;
    }

    // ��������� ������ �� ����������
    int* adev = NULL;
    hipError_t cuerr = hipMalloc((void**)&adev, n2b);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot allocate device array for a: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    int* bdev = NULL;
    cuerr = hipMalloc((void**)&bdev, n2b);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot allocate device array for b: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    int* cdev = NULL;
    cuerr = hipMalloc((void**)&cdev, n2b);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot allocate device array for c: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    // �������� ������������ �������
    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    cuerr = hipEventCreate(&start);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot create CUDA start event: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    cuerr = hipEventCreate(&stop);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot create CUDA end event: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    // ����������� ������ � ����� �� ������
    cuerr = hipMemcpy(adev, a, n2b, hipMemcpyHostToDevice);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot copy a array from host to device: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    cuerr = hipMemcpy(bdev, b, n2b, hipMemcpyHostToDevice);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot copy b array from host to device: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    // ��������� ����� ������
    cuerr = hipEventRecord(start, 0);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot record CUDA event: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    //������ ����
    for (int i = 0; i < 12; ++i) {
        kernel <<< GRID_SIZE, BLOCK_SIZE >>> (cdev, adev, bdev, n);
    }

    cuerr = hipGetLastError();
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot launch CUDA kernel: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    // ������������� ���������
    cuerr = hipDeviceSynchronize();
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot synchronize CUDA kernel: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    // ��������� ����� ���������
    cuerr = hipEventRecord(stop, 0);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot copy c array from device to host: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    // ����������� ���������� �� ����
    cuerr = hipMemcpy(c, cdev, n2b, hipMemcpyDeviceToHost);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot copy c array from device to host: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    // ������ �������
    cuerr = hipEventElapsedTime(&gpuTime, start, stop);
    printf("time spent executing %s: %.9f seconds\n", "kernel", (gpuTime / 1000) / 12);

    // �������� ������
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(adev);
    hipFree(bdev);
    hipFree(cdev);
    free(a);
    free(b);
    free(c);

    return 0;
}